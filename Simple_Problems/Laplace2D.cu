#include "hip/hip_runtime.h"
#include <cstdio>
#include "stoper.h"

/**
 * Problem is ~N, ~bandwidth limited (large size). Async send/receive suggested.
 */

/**
 * Results at Quadro2000 (PM)
 */
//Computing Laplace A; A.width=128, Kernel.width=5
//H2D copy time=  0.60[msec]
//GPU exec time=  0.14[msec]
//CPUtime=  11.88[msec]
//Computing Laplace A; A.width=256, Kernel.width=5
//H2D copy time=  0.75[msec]
//GPU exec time=  0.25[msec]
//CPUtime=  49.52[msec]
//Computing Laplace A; A.width=512, Kernel.width=5
//H2D copy time=  1.40[msec]
//GPU exec time=  0.56[msec]
//CPUtime=  231.12[msec]
//Computing Laplace A; A.width=1024, Kernel.width=5
//H2D copy time=  8.26[msec]
//GPU exec time=  2.22[msec]
//CPUtime=  958.36[msec]
//Computing Laplace A; A.width=2048, Kernel.width=5
//H2D copy time=  8.78[msec]
//GPU exec time=  7.18[msec]
//CPUtime=  3853.60[msec]
//Computing Laplace A; A.width=4096, Kernel.width=5
//H2D copy time=  32.22[msec]
//GPU exec time=  28.02[msec]
//CPUtime=  15664.01[msec]

#define REP(i,n)  for(int i=0;i<(n);++i)
typedef uint32_t uint;


//Matrices being operated on
const int kTileSize = 32;               //all problems will be tiled
const int kMatrixWidth = 8192;   //assume square matrices of this size
const int kSquareSize = kMatrixWidth * kMatrixWidth;
inline __host__ __device__ int eM(int x, int y){
  uint position = x + y*kMatrixWidth;
  position %= kMatrixWidth;
  return position;
}

const int kKernelSize = 5;
const int kBorder = kKernelSize / 2;    //separate algo for x<=border
inline __host__ __device__ int eK(int x, int y){
  uint position = x + y*kKernelSize;
  position %= kKernelSize;
  return position;
}

__global__ void Laplace(const float *A, const float *K, float *DDA);


int main(void) {
  // Allocation
  float *hA, *hDDA;              //allocated on host
  hA        = new float[kSquareSize];
  hDDA      = new float[kSquareSize];
  float LaplaceKernel[] = { 0,0,-1,0,0,  0,0,16,0,0,  -1,16,-60,16,-1,
                            0,0,-1,0,0,  0,0,16,0,0};
  REP(i, kKernelSize * kKernelSize)
    LaplaceKernel[2] /= 12.0;

  float *A, *Kernel, *DDA;                 //allocated on GPU
  hipMalloc(&A, kSquareSize * 4);
  hipMalloc(&Kernel, kKernelSize * kKernelSize * 4);
  hipMalloc(&DDA, kSquareSize * 4);
  dim3 blocks(kMatrixWidth / kTileSize, kMatrixWidth / kTileSize);
  dim3 threads(kTileSize, kTileSize);   //1024 threads; standard configuration

  printf("Computing Laplace A; A.width=%i, Kernel.width=%i\n",
      kMatrixWidth, kKernelSize);
  // Filling
  srand(12);
  for(int i=0; i<kSquareSize; ++i) {
    hA[i] = (rand() % 100 - 50) / 50.F;   //small mixed-size numbers
  }
  PosixStoper xx;
  hipMemcpy(A, hA, kSquareSize * 4, hipMemcpyHostToDevice);
  hipMemcpy(Kernel, LaplaceKernel, kKernelSize * kKernelSize * 4,
      hipMemcpyHostToDevice);
  xx.Snap(); printf("H2D copy time=\t%3.2f[msec]\n",xx.LastDt()/1000);
  Laplace<<<blocks, threads>>>(A, Kernel, DDA);
  hipDeviceSynchronize();
  xx.Snap(); printf("GPU exec time=\t%3.2f[msec]\n",xx.LastDt()/1000);
  hipMemcpy(hDDA, DDA, kSquareSize * 4, hipMemcpyDeviceToHost);

  float *htest = new float[kSquareSize];          //test results
  bzero(htest, kSquareSize * 4);
  PosixStoper yy;
  for(int x=0; x<kMatrixWidth; ++x)
    for(int y=0; y<kMatrixWidth; ++y)
    {
      for(int dx=-kBorder; dx<kBorder; ++dx)
        for(int dy=-kBorder; dy<kBorder; ++dy)
          htest[eM(x,y)] += hA[eM(x+dx,y+dy)] * LaplaceKernel[eK(dx,dy)];
    }
  yy.Snap(); printf("CPUtime=\t%3.2f[msec]\n",yy.LastDt()/1000);

  //Comparison
  int z=0;
  for(int x=0; x<kMatrixWidth; ++x)
    for(int y=0; y<kMatrixWidth; ++y)
      if (fabs(htest[eM(x,y)] - hDDA[eM(x,y)]) > 0.1) {
        ++z;
//        printf("ERR T[%i,%i]=%i\t C=%i\n", x,y, htest[x + y * kMatrixWidth],
//            hC[x + y * kMatrixWidth]);
    }
  printf("Err:%i\n",z);
  delete[] htest;
  hipFree(A); hipFree(Kernel); hipFree(DDA);
  delete[] hA; delete hDDA;
}



/**
 * Each block computes a tile (bx=column,by=row) of C.
 * It must loop over a few tiles of A and B, and sum results.
 */
__global__ void Laplace(const float *A, const float *Kernel, float *DD) {
  // Tiles held in matrices sA, sB (SMEM), loaded by threads first.
  int bx = blockIdx.x;     //block-column in C  (column in B)
  int by = blockIdx.y;     //block-row    in C  (row    in A)
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int x = bx * blockDim.x + tx;
  int y = by * blockDim.y + ty;
  DD[eM(x,y)]=0;
  for(int dx=-kBorder; dx<kBorder; ++dx)
    for(int dy=-kBorder; dy<kBorder; ++dy)
      DD[eM(x,y)] += A[eM(x+dx,y+dy)] * Kernel[eK(dx,dy)];

}
