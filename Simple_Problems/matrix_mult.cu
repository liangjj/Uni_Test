#include "hip/hip_runtime.h"
#include <cstdio>
#include "stoper.h"

/**
 * Note: CPU code scales as N^3 only for sizes up to ~128 (linux, AMD,
 *   512kB cache L2). Later dramatically slower (almost as N^4).
 */

const int kTileSize = 32;       //matrices divided into 32 x 32 tiles; CPU & GPU
const int kMatrixWidth = 256;  //assume square matrices of this size
const int kSquareSize = kMatrixWidth * kMatrixWidth;

// Single kernel for (tiled, SMEM) matrix multiplication.
__global__ void Multiply(const int *A, const int *B, int *C);
// Straightforward matrix multiplication kernel. Doesn't use shared memory SMEM.
__global__ void SlowMultiply(const int *A, const int *B, int *C);



int main(void) {
  // Allocation
  int *hA, *hB, *hC;  //allocated CPU
  hA = new int[kSquareSize];
  hB = new int[kSquareSize];
  hC = new int[kSquareSize];
  int *A, *B, *C;     //allocated on GPU
  hipMalloc(&A, kSquareSize * 4); //size in Bytes
  hipMalloc(&B, kSquareSize * 4); //size in Bytes
  hipMalloc(&C, kSquareSize * 4); //size in Bytes
  printf("Matrix multiplication; width=height=%i\n",kMatrixWidth);
  // Filling
  srand(12);
  for(int i=0; i<kSquareSize; ++i) {
    hA[i] = rand() % 100 - 50;   //small mixed-size numbers
    hB[i] = rand() % 100 - 50;   //small mixed-size numbers
  }
  // Copy to GPU
  hipMemcpy(A, hA, kSquareSize * 4, hipMemcpyHostToDevice);
  hipMemcpy(B, hB, kSquareSize * 4, hipMemcpyHostToDevice);
  // Multiply on GPU, and bring results back
  dim3 blocks(kMatrixWidth / kTileSize, kMatrixWidth / kTileSize);
  dim3 threads(kTileSize, kTileSize);
  PosixStoper xx;
  Multiply<<<blocks, threads>>>(A, B, C);
  hipMemcpy(hC, C, kSquareSize * 4, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  xx.Snap(); printf("GPUtime=%3.2f[msec]\n",xx.LastDt()/1000);
  // Test result on CPU
  int *htest = new int[kSquareSize];  //tested on CPU
  bzero(htest, kSquareSize * 4);  //clean it up
  PosixStoper yy;
  int tmp=kMatrixWidth;
  for(int x=0; x<tmp; ++x)
    for(int y=0; y<tmp; ++y)
      for(int i=0; i<tmp; ++i)
        htest[x + y *kMatrixWidth] += hA[i + y * kMatrixWidth] *
                                      hB[x + i * kMatrixWidth];
  yy.Snap(); printf("CPUtime=%3.2f[msec]\n",yy.LastDt()/1000);

  //Comparison
  int z=0;
  for(int x=0; x<kMatrixWidth; ++x)
    for(int y=0; y<kMatrixWidth; ++y)
      if (htest[x + y * kMatrixWidth] != hC[x + y * kMatrixWidth])
        ++z;
//        printf("ERR T[%i,%i]=%i\t C=%i\n", x,y, htest[x + y * kMatrixWidth],
//            hC[x + y * kMatrixWidth]);
  printf("Err:%i\n",z);
  delete[] htest;
  hipFree(A); hipFree(B); hipFree(C);
  delete[] hA; delete[] hB; delete hC;
}





/**
 * Each block computes a tile (bx=column,by=row) of C.
 * It must loop over a few tiles of A and B, and sum results.
 */
__global__ void Multiply(const int *A, const int *B, int *C) {
  // Tiles held in matrices sA, sB (SMEM), loaded by threads first.
  int bx = blockIdx.x;     //block-column in C  (column in B)
  int by = blockIdx.y;     //block-row    in C  (row    in A)
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int bk;   //index for loop over block of _tiles_ in A (row) and B (column).
  int Csub = 0;  //Store locally data in loop; write to GMEM only once at end.
  __shared__ float sA[kTileSize][kTileSize];       //"tile" matrices
  __shared__ float sB[kTileSize][kTileSize];
  // Loop over tiles, for each block in C seleceted by (bx,by)
  for(bk=0; bk < kMatrixWidth / kTileSize; ++bk) {
    //load matrices into SMEM
    sA[ty][tx] = A[(by * kTileSize + ty) * kMatrixWidth + (bk * kTileSize) +tx];
    sB[ty][tx] = B[(bk * kTileSize + ty) * kMatrixWidth + (bx * kTileSize) +tx];
    __syncthreads();
    // Multiple the tiles A * B --store--> C
    for(int k=0; k<kTileSize; ++k)
      Csub += sA[ty][k] * sB[k][tx];
    __syncthreads();
  }
  C[(by * kTileSize + ty) * kMatrixWidth + (bx * kTileSize + tx)] = Csub;
}


/**
 * Each block computes a tile (bx=column, by=row) of C.
 * It must loop over a few tiles of A and B, and sum results.
 */
__global__ void SlowMultiply(const int *A, const int *B, int *C) {
  int bx = blockIdx.x;     //block-column in C  (column in B)
  int by = blockIdx.y;     //block-row    in C  (row    in A)
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int bk;   //index for loop over block of _tiles_ in A (row) and B (column).
  int Csub = 0;
  // Loop over tiles, for each block in C seleceted by (bx,by)
  for(bk=0; bk < kMatrixWidth / kTileSize; ++bk) {
    for(int k=0; k<kTileSize; ++k) {  //loop over index in the tile
      Csub +=  A[(by * kTileSize + ty) * kMatrixWidth + (bk * kTileSize) + k]
             * B[(bk * kTileSize + k) * kMatrixWidth + (bx * kTileSize) + tx];
    }
    __syncthreads();
  }
  C[(by * kTileSize + ty) * kMatrixWidth + (bx * kTileSize + tx)] = Csub;
}





